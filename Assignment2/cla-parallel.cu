#include "hip/hip_runtime.h"
/*********************************************************************/
//
// 02/02/2023: Revised Version for 32M bit adder with 32 bit blocks
//
/*********************************************************************/

#include "main.h"

//Touch these defines
#define input_size 8388608 // hex digits 
#define block_size 32
#define verbose 0

//Do not touch these defines
#define digits (input_size+1)
#define bits (digits*4)
#define ngroups bits/block_size
#define nsections ngroups/block_size
#define nsupersections nsections/block_size
#define nsupersupersections nsupersections/block_size

//Global definitions of the various arrays used in steps for easy access
/***********************************************************************************************************/
// ADAPT AS CUDA managedMalloc memory - e.g., change to pointers and allocate in main function. 
/***********************************************************************************************************/
int* gi;
int* pi; 
int* ci;

int* ggj;
int* gpj;
int* gcj;

int* sgk;
int* spk;
int* sck;

int* ssgl;
int* sspl;
int* sscl;

int* ssspm;
int* sssgm;
int* ssscm;

int* dsumi;
int* dbin1;
int* dbin2;

//host side
int sumi[bits] = {0};
int sumrca[bits] = {0};

//Integer array of inputs in binary form
int* bin1=NULL;
int* bin2=NULL;

//Character array of inputs in hex form
char* hex1=NULL;
char* hex2=NULL;

void read_input()
{
	char* in1 = (char *)calloc(input_size+1, sizeof(char));
	char* in2 = (char *)calloc(input_size+1, sizeof(char));

	if( 1 != scanf("%s", in1))
	{
		printf("Failed to read input 1\n");
		exit(-1);
	}
	if( 1 != scanf("%s", in2))
	{
		printf("Failed to read input 2\n");
		exit(-1);
	}
	
	hex1 = grab_slice_char(in1,0,input_size+1);
	hex2 = grab_slice_char(in2,0,input_size+1);
	
	free(in1);
	free(in2);
}

/***********************************************************************************************************/
// ADAPT AS CUDA KERNEL 
/***********************************************************************************************************/

//This function computes the gs and ps for all bits
__global__ void compute_gp(const int* b1, const int* b2, int* g, int* p){
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i>=bits) 
		return; //Throw out any aditional threads 
	g[i] = b1[i] & b2[i];
	p[i] = b1[i] | b2[i];
}

//general kernal used to calculate the gp values for every level other than the lowest one. the 
//formula for a group of 4 bits is as follows: 
//ggj = gi+3 + pi+3gi+2 + pi+3pi+2gi+1 + pi+3pi+2pi+1gi
//gpj = pi+3pi+2pi+1pi 
//this can be expanded to calculate the values for any given block size
__global__ void compute_general_gp(int nbit, const int* prevG, const int* prevP, int* curG, int* curP){
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i>=nbit) 
		return; //Throw out any aditional threads 
	int start = i*block_size;

	//pointers to the groups that the thread is reducing
	const int* g_group = prevG+start;
	const int* p_group = prevP+start;
	
	//calculating g values
	int sum = 0;
	for(int j = 0; j < block_size; j++){
		int mult = g_group[j]; //grabs the g_i term for the multiplication
		for(int k = block_size-1; k > j; k--) {
			mult &= p_group[k]; //grabs the p_i terms and multiplies it with the previously multiplied stuff (or the g_i term if first round)
		}
		sum |= mult; //sum up each of these things with an or
	}
	curG[i] = sum;

	//calculating p values
	int mult = p_group[0];
	for(int j = 1; j < block_size; j++) {
		mult &= p_group[j];
	}
	curP[i] = mult;
}

//calls general compute gp function with appropriate arguments for given level
void compute_group_gp(){
	compute_general_gp<<<ngroups,32>>>(ngroups, gi, pi, ggj, gpj);
}

//calls general compute gp function with appropriate arguments for given level
void compute_section_gp() {
	compute_general_gp<<<nsections,32>>>(nsections, ggj, gpj, sgk, spk);
}

//calls general compute gp function with appropriate arguments for given level
void compute_super_section_gp() {
	compute_general_gp<<<nsupersections,32>>>(nsupersections, sgk, spk, ssgl, sspl);
}

void compute_super_super_section_gp() {
	compute_general_gp<<<nsupersupersections,32>>>(nsupersupersections, ssgl, sspl, sssgm, ssspm);
}

//general function to compute the carry bits for the current "level". takes the number of sections/groups, the current 
//level's g array, p array c array and the "above" level's c array
__global__ void compute_general_carry(int n, const int* prevG, const int* prevP, const int* curC, int* prevC){
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i>=n) return;
	int carry = i == 0 ? 0 : curC[i-1];
	for(int j = i*block_size; j < (i+1)*block_size; j++){
		prevC[j] = prevG[j] | (prevP[j] & carry);
		carry = prevC[j];
	}
}

//calls general compute carry function with appropriate arguments for given level
void compute_super_super_section_carry()
{
	compute_general_carry<<<1,32>>>(1, sssgm, ssspm, nullptr, ssscm);
}

//calls general compute carry function with appropriate arguments for given level
void compute_super_section_carry()
{
	compute_general_carry<<<nsupersupersections,32>>>(nsupersupersections, ssgl, sspl, ssscm, sscl);
}

//calls general compute carry function with appropriate arguments for given level
void compute_section_carry()
{
	compute_general_carry<<<nsupersections,32>>>(nsupersections, sgk, spk, sscl, sck);
}

//calls general compute carry function with appropriate arguments for given level
void compute_group_carry()
{
	compute_general_carry<<<nsections,32>>>(nsections, ggj, gpj, sck, gcj);
}

//calls general compute carry function with appropriate arguments for given level
void compute_carry()
{
	compute_general_carry<<<ngroups,32>>>(ngroups, gi, pi, gcj, ci);
}

__global__ void compute_sum(int* sum, const int* b1, const int* b2, const int* c){
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i>=bits) return;
	int carry = i == 0 ? 0 : c[i-1];
	if (i==0) {carry = 0;}
	else {carry = c[i-1];}
	sum[i] = b1[i] ^ b2[i] ^ carry;
}

void cla()
{
	//allocating all the memory on the GPU
	hipMallocManaged((void**)&gi, bits*sizeof(int));
	hipMallocManaged((void**)&pi, bits*sizeof(int));
	hipMallocManaged((void**)&ci, bits*sizeof(int));
	hipMallocManaged((void**)&ggj, ngroups*sizeof(int));
	hipMallocManaged((void**)&gpj, ngroups*sizeof(int));
	hipMallocManaged((void**)&gcj, ngroups*sizeof(int));
	hipMallocManaged((void**)&sgk, nsections*sizeof(int));
	hipMallocManaged((void**)&spk, nsections*sizeof(int));
	hipMallocManaged((void**)&sck, nsections*sizeof(int));
	hipMallocManaged((void**)&ssgl, nsupersections*sizeof(int));
	hipMallocManaged((void**)&sspl, nsupersections*sizeof(int));
	hipMallocManaged((void**)&sscl, nsupersections*sizeof(int));
	hipMallocManaged((void**)&sssgm, nsupersupersections*sizeof(int));
	hipMallocManaged((void**)&ssspm, nsupersupersections*sizeof(int));
	hipMallocManaged((void**)&ssscm, nsupersupersections*sizeof(int));

	//getting the gp values for the lowest level
	compute_gp<<<bits,32>>>(dbin1, dbin2, gi, pi);

	//each of these functions will call the general versions which will take care of everything
	//computing gps for each level
	compute_group_gp();
	compute_section_gp();
	compute_super_section_gp();
	compute_super_super_section_gp();

	//computing carries for each level
	compute_super_super_section_carry();
	compute_super_section_carry();
	compute_section_carry();
	compute_group_carry();
	compute_carry();

	//computing the sum
	compute_sum<<<bits,32>>>(dbin1, dbin2, ci, dsumi);

	hipDeviceSynchronize();
}

void ripple_carry_adder()
{
	int clast=0, cnext=0;

	for(int i = 0; i < bits; i++)
		{
			cnext = (bin1[i] & bin2[i]) | ((bin1[i] | bin2[i]) & clast);
			sumrca[i] = bin1[i] ^ bin2[i] ^ clast;
			clast = cnext;
		}
}

void check_cla_rca()
{
	for(int i = 0; i < bits; i++)
		{
			if( sumrca[i] != sumi[i] )
	{
		printf("Check: Found sumrca[%d] = %d, not equal to sumi[%d] = %d - stopping check here!\n",
		 i, sumrca[i], i, sumi[i]);
		printf("bin1[%d] = %d, bin2[%d]=%d, gi[%d]=%d, pi[%d]=%d, ci[%d]=%d, ci[%d]=%d\n",
		 i, bin1[i], i, bin2[i], i, gi[i], i, pi[i], i, ci[i], i-1, ci[i-1]);
		return;
	}
		}
	printf("Check Complete: CLA and RCA are equal\n");
}

int main(int argc, char *argv[])
{
	int randomGenerateFlag = 1;
	int deterministic_seed = (1<<30) - 1;
	char* hexa=NULL;
	char* hexb=NULL;
	char* hexSum=NULL;
	char* int2str_result=NULL;
	unsigned long long start_time=clock_now(); // dummy clock reads to init
	unsigned long long end_time=clock_now();   // dummy clock reads to init

	if( nsupersupersections != block_size )
		{
			printf("Misconfigured CLA - nsupersupersections (%d) not equal to block_size (%d) \n",
			 nsupersupersections, block_size );
			return(-1);
		}
	
	if (argc == 2) {
		if (strcmp(argv[1], "-r") == 0)
			randomGenerateFlag = 1;
	}
	
	if (randomGenerateFlag == 0)
		{
			read_input();
		}
	else
		{
			srand( deterministic_seed );
			hex1 = generate_random_hex(input_size);
			hex2 = generate_random_hex(input_size);
		}
	
	hexa = prepend_non_sig_zero(hex1);
	hexb = prepend_non_sig_zero(hex2);
	hexa[digits] = '\0'; //double checking
	hexb[digits] = '\0';
	
	bin1 = gen_formated_binary_from_hex(hexa);
	bin2 = gen_formated_binary_from_hex(hexb);

	start_time = clock_now();
	cla();
	end_time = clock_now();

	printf("CLA Completed in %llu cycles\n", (end_time - start_time));

	start_time = clock_now();
	ripple_carry_adder();
	end_time = clock_now();

	printf("RCA Completed in %llu cycles\n", (end_time - start_time));

	check_cla_rca();

	if( verbose==1 )
		{
			int2str_result = int_to_string(sumi,bits);
			hexSum = revbinary_to_hex( int2str_result,bits);
		}

	// free inputs fields allocated in read_input or gen random calls
	free(int2str_result);
	free(hex1);
	free(hex2);
	
	// free bin conversion of hex inputs
	free(bin1);
	free(bin2);
	
	if( verbose==1 )
		{
			printf("Hex Input\n");
			printf("a   ");
			print_chararrayln(hexa);
			printf("b   ");
			print_chararrayln(hexb);
		}
	
	if ( verbose==1 )
		{
			printf("Hex Return\n");
			printf("sum =  ");
		}
	
	// free memory from prepend call
	free(hexa);
	free(hexb);

	if( verbose==1 )
		printf("%s\n",hexSum);
	
	free(hexSum);
	
	return 0;
}
